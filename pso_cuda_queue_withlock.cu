#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>
#include "common.h"
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

// cuda prototype
__host__ __device__ double fit(double x);
__global__ void move(double *position_d, double *velocity_d, double *fitness_d, 
    double *pbest_pos_d, double *pbest_fit_d, volatile particle *gbest, int *lock);

//cuda constant memory
__constant__ double w_d;
__constant__ double c1_d;
__constant__ double c2_d;
__constant__ double max_pos_d;
__constant__ double min_pos_d;
__constant__ double max_v_d;
__constant__ int max_iter_d;
__constant__ int particle_cnt_d;
__constant__ int tile_size;
__constant__ int tile_size2;

//cuda function
__global__ void move(double *position_d, double *velocity_d, double *fitness_d, 
    double *pbest_pos_d, double *pbest_fit_d, volatile particle *gbest, int *lock){
    int idx =  blockIdx.x * blockDim.x + threadIdx.x;
    int tidx = threadIdx.x;
    extern __shared__ double sharedMemory[];
    double *privateBestQueue    = (double *)sharedMemory;
    double *privateBestPosQueue = (double *)&sharedMemory[tile_size];
    __shared__ unsigned int queue_num;
    double v    = velocity_d[idx];
    double pos  = position_d[idx];
    double ppos = pbest_pos_d[idx];
    double fitness  = fitness_d[idx];
    double pfitness = pbest_fit_d[idx];
    hiprandState state1, state2;
    hiprand_init((unsigned long long)clock() + idx, 0, 0, &state1);
    hiprand_init((unsigned long long)clock() + idx, 0, 0, &state2);
    if(idx < particle_cnt_d){
        if(tidx == 0)
            queue_num = 0;
        v = w_d * v + c1_d * hiprand_uniform_double(&state1) * (ppos - pos) 
            + c2_d * hiprand_uniform_double(&state2) * (gbest->position - pos);
        if(v < -max_v_d)
            v = -max_v_d;
        else if(v > max_v_d)
            v = max_v_d;
        pos = pos + v;
        if(pos > max_pos_d)
            pos = max_pos_d; // 限制最大位置
        else if(pos < min_pos_d)
            pos = min_pos_d; // 限制最小位置
        fitness = fit(pos);
        if(fitness > pfitness){
            pbest_pos_d[idx] = pos;
            pbest_fit_d[idx] = fitness;
        }
        privateBestPosQueue[0] = INT_MIN;
        privateBestQueue[0] = INT_MIN;
    }
    __syncthreads();
    if(fitness > gbest->fitness){
        unsigned const my_index = atomicAdd(&queue_num, 1);
        privateBestPosQueue[my_index] = pos;
        privateBestQueue[my_index] = fitness;
    }
    __syncthreads();
    if(idx < particle_cnt_d){
        if(tidx==0){
            if(queue_num){
                for(int j=1; j<queue_num; j++){
                    if(privateBestQueue[j] > privateBestQueue[0]){
                        privateBestPosQueue[0] = privateBestPosQueue[j];
                        privateBestQueue[0] = privateBestQueue[j];
                    }
                }
                while(atomicCAS(lock, 0, 1) != 0);
                if(privateBestQueue[0] > gbest->fitness){
                    gbest->position = privateBestPosQueue[0];
                    gbest->fitness  = privateBestQueue[0];
                    __threadfence();
                }
                atomicExch(lock, 0);
            }
        }
        position_d[idx] = pos;
        velocity_d[idx] = v;
        fitness_d[idx]  = fitness;
    }
}

__host__ __device__ double fit(double x){
    // x**3 - 0.8x**2 - 1000x + 8000
    return fabs(8000.0 + x*(-10000.0+x*(-0.8+x)));
}

int main(int argc, char **argv){
    arguments args = {100000, 4096, 1024, 4, 0};
    int retError = pargeArgs(&args, argc, argv);
    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    float exe_time;
    // 變數宣告
    clock_t begin_app  = clock();
    clock_t begin_init = begin_app;
    particle_Coal *p; // p : 粒子群

    double *position_d;
    double *velocity_d;
    double *fitness_d;
    double *pbest_pos_d;
    double *pbest_fit_d;
    particle *gbest_d;
    int *lock_d; // block level lock for gbest
    int block_size = min(1024, args.blocks_per_grid);

    // 設定參數
    min_pos = -100.0 , max_pos = +100.0;  // 位置限制, 即解空間限制
    w = 1, c1 = 2.0, c2 = 2.0;            // 慣性權重與加速常數設定
    particle_cnt = args.particle_cnt;     // 設粒子個數
    max_v = (max_pos-min_pos) * 1.0;      // 設最大速限

    p = (particle_Coal*) malloc(sizeof(particle_Coal));
    //p->position  = (double *) malloc(sizeof(double)* particle_cnt);
    //p->velocity  = (double *) malloc(sizeof(double)* particle_cnt); 
    //p->fitness   = (double *) malloc(sizeof(double)* particle_cnt); 
    //p->pbest_pos = (double *) malloc(sizeof(double)* particle_cnt); 
    //p->pbest_fit = (double *) malloc(sizeof(double)* particle_cnt);
    ParticleInitCoal(p); // 粒子初始化

    printf("Allocating device memory\n");
    //HANDLE_ERROR(hipMalloc((void **)&p_d, sizeof(particle_Coal)));
    HANDLE_ERROR(hipMalloc((void **)&position_d, sizeof(double)* particle_cnt));
    HANDLE_ERROR(hipMalloc((void **)&velocity_d, sizeof(double)* particle_cnt));
    HANDLE_ERROR(hipMalloc((void **)&fitness_d, sizeof(double)* particle_cnt));
    HANDLE_ERROR(hipMalloc((void **)&pbest_pos_d, sizeof(double)* particle_cnt));
    HANDLE_ERROR(hipMalloc((void **)&pbest_fit_d, sizeof(double)* particle_cnt));
    HANDLE_ERROR(hipMalloc((void **)&gbest_d, sizeof(particle)));
    HANDLE_ERROR(hipMalloc((void**)&lock_d, sizeof(int)));
    printf("Copying to device\n");
    HANDLE_ERROR(hipMemcpy(position_d, p->position, sizeof(double)* particle_cnt, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(velocity_d, p->velocity, sizeof(double)* particle_cnt, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(fitness_d, p->fitness, sizeof(double)* particle_cnt, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(pbest_pos_d, p->pbest_pos, sizeof(double)* particle_cnt, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(pbest_fit_d, p->pbest_fit, sizeof(double)* particle_cnt, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(gbest_d, &gbest, sizeof(particle), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(w_d), &w, sizeof(double)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(c1_d), &c1, sizeof(double)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(c2_d), &c2, sizeof(double)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(max_pos_d), &max_pos, sizeof(double)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(min_pos_d), &min_pos, sizeof(double)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(max_v_d), &max_v, sizeof(double)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(max_iter_d), &args.max_iter, sizeof(int)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(particle_cnt_d), &args.particle_cnt, sizeof(int)));
    //HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(tile_size), &args.block_queue_size, sizeof(int)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(tile_size), &args.threads_per_block, sizeof(int)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(tile_size2), &block_size, sizeof(int)));
    HANDLE_ERROR(hipMemset(lock_d, 0, sizeof(int)));
    clock_t end_init = clock();
    clock_t begin_exe  = end_init;
    HANDLE_ERROR(hipEventRecord(start));
    for(unsigned int i = 0; i < args.max_iter; i++){
        move<<<args.blocks_per_grid, args.threads_per_block, sizeof(double) * (2 * args.threads_per_block + 1)>>>
            (position_d, velocity_d, fitness_d, pbest_pos_d, pbest_fit_d, 
                gbest_d, lock_d);
    }
    HANDLE_ERROR(hipEventRecord(stop));
    HANDLE_ERROR(hipMemcpy(p->position, position_d, sizeof(double)* particle_cnt, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(p->velocity, velocity_d, sizeof(double)* particle_cnt, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(p->fitness, fitness_d, sizeof(double)* particle_cnt, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(p->pbest_pos, pbest_pos_d, sizeof(double)* particle_cnt, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(p->pbest_fit, pbest_fit_d, sizeof(double)* particle_cnt, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(&gbest, gbest_d, sizeof(particle), hipMemcpyDeviceToHost));
    clock_t end_exe  = clock();
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&exe_time, start, stop));

    //for(int i=0; i<particle_cnt; i++)
    //    printf("#%d : %lf , %lf . %lf\n", i+1, p->position[i], p->fitness[i], p->velocity[i]);
    free(p);
    hipFree(position_d);
    hipFree(velocity_d);
    hipFree(fitness_d);
    hipFree(pbest_pos_d);
    hipFree(pbest_fit_d);
    hipFree(gbest_d);
    hipFree(lock_d);
    printf("the answer : %10.6lf, %lf\n", -57.469, fit(-57.469));
    printf("best result: %10.6lf, %lf\n", gbest.position, gbest.fitness);
    printf("[Initial   time]: %lf (sec)\n", (double)(end_init - begin_init) / CLOCKS_PER_SEC);
    //printf("[Execution time]: %lf (sec)\n", (double)(end_exe - begin_exe) / CLOCKS_PER_SEC);
    printf("[Cuda Exec time]: %f (sec)\n", exe_time / 1000);
    printf("[Elapsed   time]: %lf (sec)\n", (double)(clock() - begin_app) / CLOCKS_PER_SEC);
    return 0;
}

void ParticleInitCoal(particle_Coal *p){
	unsigned int i;
	const double pos_range = max_pos - min_pos; // 解寬度
    srand((unsigned)time(NULL));
    p->position  = (double *) malloc(sizeof(double)* particle_cnt);
    p->velocity  = (double *) malloc(sizeof(double)* particle_cnt); 
    p->fitness   = (double *) malloc(sizeof(double)* particle_cnt); 
    p->pbest_pos = (double *) malloc(sizeof(double)* particle_cnt); 
    p->pbest_fit = (double *) malloc(sizeof(double)* particle_cnt);
	// 以下程式碼效率不佳, 但較易懂一點
	for(i=0; i<particle_cnt; i++) {
		// 隨機取得粒子位置, 並設為該粒子目前最佳適應值
		p->pbest_pos[i] = p->position[i] = RND() * pos_range + min_pos; 
		// 隨機取得粒子速度
		p->velocity[i] = RND() * max_v;
		// 計算該粒子適應值, 並設為該粒子目前最佳適應值
		p->pbest_fit[i] = p->fitness[i] = fit(p->position[i]);
		// 全域最佳設定
		if(i==0 || p->pbest_fit[i] > gbest.fitness){
			gbest.position = p->position[i];      // 目前位置, 即x value    
			gbest.velocity = p->velocity[i];      // 目前粒子速度           
			gbest.fitness = p->fitness[i];       // 適應函式值            
			gbest.pbest_pos = p->pbest_pos[i];    // particle 目前最好位置
			gbest.pbest_fit = p->pbest_fit[i];   // particle 目前最佳適應值
		} 
    }
}
